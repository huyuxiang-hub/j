#include "hip/hip_runtime.h"
#include "stdio.h"
#include <cassert>
#include "Layr.h"
#include "LayrTest.h"

#define LAYRTEST_API  __attribute__ ((visibility ("default")))

template<typename T, int N>
__global__ void Stack_computeART_gpu(LayrTestData<T,N>* ltd, const StackSpec<T> spec )
{ 
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= ltd->ni) return ;  
 
    const T th = ltd->theta[idx] ; 
    //printf("// ltd.ni %d idx %d spec.n1i %10.4f th %10.4f   \n", ltd->ni, idx, spec.n1i, th ); 

    Stack<T,N> stk(ltd->wl, th, spec) ; 

    ltd->arts[idx] = stk.art ; 
    ltd->comps[idx] = stk.comp ; 
    for(int l=0 ; l < N ; l++) ltd->lls[N*idx+l] = stk.ll[l]; 

    //printf("// ltd.ni %d idx %d stk.art.R %10.4f stk.art.T %10.4f  \n", ltd->ni, idx, stk.art.R, stk.art.T ); 
} 

void ConfigureLaunch(dim3& numBlocks, dim3& threadsPerBlock, unsigned width )
{ 
    threadsPerBlock.x = 512 ; 
    threadsPerBlock.y = 1 ; 
    threadsPerBlock.z = 1 ; 

    numBlocks.x = (width + threadsPerBlock.x - 1) / threadsPerBlock.x ; 
    numBlocks.y = 1 ; 
    numBlocks.z = 1 ; 
}

template<typename T, int N>
void LayrTest_launch(LayrTest<T,N>& t, const StackSpec<T>& spec )
{
    //printf("[LayrTest_launch t.d_ptr %p \n", t.d_ptr ); 
    assert( t.d_ptr ); 
    dim3 numBlocks ; 
    dim3 threadsPerBlock ; 
    ConfigureLaunch(numBlocks, threadsPerBlock, t.d.ni );  
    Stack_computeART_gpu<T,N><<<numBlocks,threadsPerBlock>>>(t.d_ptr, spec) ; 
    //printf("]LayrTest_launch t.d_ptr %p \n", t.d_ptr ); 
}

template LAYRTEST_API void LayrTest_launch(LayrTest<float, 4>&, const StackSpec<float>&  ); 
template LAYRTEST_API void LayrTest_launch(LayrTest<double,4>&, const StackSpec<double>&  ); 

