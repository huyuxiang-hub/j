#include "hip/hip_runtime.h"
#include "stdio.h"
#include "Layr.h"
#include "LayrTest.h"

__global__ 
void on_device(LayrTestData<float,4>* ltd)
{ 
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    if( idx >= ltd->ni ) return ;  

    float th = ltd->theta[idx] ; 
    StackSpec<float> spec = StackSpec<float>::Default() ; 
    Stack<float,4> stk(ltd->wl, spec) ; 
    stk.computeART(th); 

    printf("// ltd.ni %d idx %d stk.art.R %10.4f stk.art.T %10.4f  \n", ltd->ni, idx, stk.art.R, stk.art.T ); 

    ltd->arts[idx] = stk.art ; 
    ltd->comps[idx] = stk.comp ; 
    for(int l=0 ; l < 4 ; l++) ltd->lls[4*idx+l] = stk.ll[l]; 
} 

void ConfigureLaunch(dim3& numBlocks, dim3& threadsPerBlock, unsigned width )
{ 
    threadsPerBlock.x = 512 ; 
    threadsPerBlock.y = 1 ; 
    threadsPerBlock.z = 1 ; 

    numBlocks.x = (width + threadsPerBlock.x - 1) / threadsPerBlock.x ; 
    numBlocks.y = 1 ; 
    numBlocks.z = 1 ; 
}


#ifdef WITH_CUDA_LAUNCH_MAIN
int main()
{
    LayrTest<float,4> t ; 

    t.upload(); 
    assert( t.d_ptr ); 

    dim3 numBlocks ; 
    dim3 threadsPerBlock ; 
    ConfigureLaunch(numBlocks, threadsPerBlock, t.d.ni );  

    on_device<<<numBlocks,threadsPerBlock>>>(t.d_ptr) ; 
    hipDeviceSynchronize();

    t.download();

    t.save("/tmp/LayrTest2"); 

    return 0 ; 
}
#endif



